#include <iostream>
#include <iomanip>
#include <string>

#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/version.hpp>

#include"interface.h"
#include"compute.h"




static U8 *cuda_l;


void interface(cv::Mat left, cv::Mat right)
{
  

   cv::Mat I1 = left(cv::Rect(0, 0, 320, 320));
   cv::Mat I2 = right(cv::Rect(0, 0, 320, 320));

	cv::imshow("left",I1);


   const int width = I1.cols;
   const int height = I1.rows;
   std::cout<<width<<" ,"<<height<<std::endl;
   
   const int size = width * height;

   std::cout<<"compute"<<std::endl;
	hipMalloc((void **)&cuda_l, sizeof(U8)*size);


//	compute(left, count);

	cv::waitKey(0);

   

}

